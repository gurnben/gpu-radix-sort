#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#include <iostream>
#include <ctime>

#include "sort.h"
#include "utils.h"

double test_cpu_vs_gpu(unsigned int* h_in, unsigned int num_elems)
{
    std::clock_t start;

    unsigned int* h_out_cpu = new unsigned int[num_elems];
    unsigned int* h_out_gpu = new unsigned int[num_elems];

    unsigned int* d_in;
    unsigned int* d_out;
    checkCudaErrors(hipMalloc(&d_in, sizeof(unsigned int) * num_elems));
    checkCudaErrors(hipMalloc(&d_out, sizeof(unsigned int) * num_elems));
    checkCudaErrors(hipMemcpy(d_in, h_in, sizeof(unsigned int) * num_elems, hipMemcpyHostToDevice));
    start = std::clock();
    radix_sort(d_out, d_in, num_elems);
    double gpu_duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
    checkCudaErrors(hipMemcpy(h_out_gpu, d_out, sizeof(unsigned int) * num_elems, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_out));
    checkCudaErrors(hipFree(d_in));

    delete[] h_out_gpu;
    delete[] h_out_cpu;

    return gpu_duration;
}

/*printUsage
*
* Prints the usage information for this application.
*/
void printUsage()
{
    printf("This application takes as input a lower and upper bound for\n");
    printf("data sizes.  The input lower and upper bounds are taken in as\n");
    printf("powers of 2.  For example, input 24 as a lower bound will sort\n");
    printf("2^24 elements.\n");
    printf("\nusage: radix_sort <lower bound> <upper bound>\n");
    printf("\t<lower_bound> will be treated as a power of 2 and is inclusive\n");
    printf("\t<upper_bound> will be treated as a power of 2 and is inclusive\n");
    printf("Examples:\n");
    printf("\t./hybrid_sort 24 26\n");
}

/*parseCommandArgs
*
* This function processes the command line arguments given to the program.
*
* the proper use is:
*   ./hybrid_sort <lower_bound> <upper_bound>
*
* @params:
*   argc        - the number of arguments in argv
*   argv        - the arguments to the utility
*   lower_bound - a pointer to a lower_bound variable
*   upper_bound - a pointer to an upper_bound variable
*/
void parseCommandArgs(int argc, char * argv[], int * lower_bound,
                      int * upper_bound) {
    if (argc < 3) {
      printUsage();
      //exit because the input was incorrect
      exit(EXIT_FAILURE);
    }
    else {
      (*lower_bound) = atoi(argv[argc - 2]);
      (*upper_bound) = atoi(argv[argc - 1]);
    }
}

int main(int argc, char * argv[])
{
    // Set up clock for timing comparisons
    srand(1);

    int lower_bound = 0, upper_bound = 0;

    parseCommandArgs(argc, argv, &lower_bound, &upper_bound);

    for (int i = lower_bound; i <= upper_bound; ++i)
    {
        unsigned int num_elems = (1 << i);
        unsigned int* h_in = new unsigned int[num_elems];
        unsigned int* h_in_rand = new unsigned int[num_elems];

        for (unsigned int j = 0; j < num_elems; j++)
        {
            h_in[j] = (num_elems - 1) - j;
            h_in_rand[j] = rand() % num_elems;
        }
        double time = 0;
        for (unsigned int j = 0; j < 2; ++j) {
            time = test_cpu_vs_gpu(h_in_rand, num_elems);
        }

        printf("Four-way Radix Sort took %f milliseconds to sort %e (2^%d) numbers.\n", time, pow(2, i), i);

        delete[] h_in;
        delete[] h_in_rand;
    }
}
